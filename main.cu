#include "hip/hip_runtime.h"
#include "camera.cuh"
#include "material.cuh"
#include "pngmaster.h"
#include "ray.cuh"
#include "scene.cuh"
#include "shpere.cuh"
#include "vector3.cuh"
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <iostream>
#include <stdio.h>
#include <time.h>
// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func,
                const char *const file, int const line) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at "
              << file << ":" << line << " '" << func << "' \n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}

#define RANDVEC3                                                               \
  vector3(hiprand_uniform(local_rand_state), hiprand_uniform(local_rand_state),  \
          hiprand_uniform(local_rand_state))

__device__ vector3 color(const ray &_r, scene **_tmp_scene,
                         hiprandState *local_rand_state) {

  ray cur_ray = _r;
  vector3 cur_attenuation(1.0f, 1.0f, 1.0f);
  for (int i = 0; i < 20; ++i) {
    hitinfo tmp_info;
    if ((*_tmp_scene)->hit(cur_ray, 0.001f, FLT_MAX, tmp_info)) {
      ray scattered_ray;
      vector3 attenutation;
      if (tmp_info.material_ptr->scatter(cur_ray, tmp_info, attenutation,
                                         scattered_ray, local_rand_state)) {
        cur_attenuation *= attenutation;
        cur_ray = scattered_ray;
      } else {
        return vector3(0.0f, 0.0f, 0.0f);
      }
    } else {
      vector3 unit_direction = _r.direction().normalize();
      float t = 0.5f * (unit_direction.y() + 1.0f);
      return cur_attenuation * (1.0f - t) * vector3(1, 1, 1) +
             t * vector3(0.5f, 0.7f, 1.0f);
    }
  }
  return vector3(0.0f, 0.0f, 0.0f);
}

__global__ void render(vector3 *fb, int max_x, int max_y, int ray_num,
                       camera **tmp_cam, scene **tmp_scene,
                       hiprandState *rand_state) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ((i >= max_x) || (j >= max_y))
    return;
  int pixel_index = j * max_x + i;
  hiprandState local_rand_state = rand_state[pixel_index];

  vector3 tmp_col(0, 0, 0);
  for (int r = 0; r < ray_num; ++r) {
    float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
    float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
    ray tmp_r = (*tmp_cam)->gen_ray(u, v, rand_state);
    tmp_col += color(tmp_r, tmp_scene, &local_rand_state);
  }
  rand_state[pixel_index] = local_rand_state;
  tmp_col /= float(ray_num);
  // tmp_col[0] = sqrt(tmp_col[0]);
  // tmp_col[1] = sqrt(tmp_col[1]);
  // tmp_col[2] = sqrt(tmp_col[2]);
  fb[pixel_index] = tmp_col;
}
__global__ void rand_init(int max_x, int max_y, hiprandState *rand_state) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ((i >= max_x) || (j >= max_y))
    return;
  int pixel_index = j * max_x + i;
  // Each thread gets same seed, a different sequence number, no offset
  hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}
__global__ void init_scene(object **objs, scene **tmp_scene, camera **tmp_cam,
                           int nx, int ny, hiprandState *rand_state) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    hiprandState local_rand_state = *rand_state;
    objs[0] = new sphere(vector3(0.0f, -1000.0f, -1.0f), 1000.0f,
                         new lambertian(vector3(0.5f, 0.5f, 0.5f)));
    int i = 1;
    for (int a = -11; a < 11; ++a) {
      for (int b = -11; b < 11; ++b) {
        float choose_mat = (hiprand_uniform(&local_rand_state));
        vector3 tmp_center(a + hiprand_uniform(&local_rand_state), 0.2f,
                           b + hiprand_uniform(&local_rand_state));
        if (choose_mat < 0.8f) {
          objs[i++] = new sphere(
              tmp_center, 0.2f,
              new lambertian(vector3(hiprand_uniform(&local_rand_state) *
                                         hiprand_uniform(&local_rand_state),
                                     hiprand_uniform(&local_rand_state) *
                                         hiprand_uniform(&local_rand_state),
                                     hiprand_uniform(&local_rand_state) *
                                         hiprand_uniform(&local_rand_state))));

        } else if (choose_mat < 0.95f) {
          objs[i++] = new sphere(
              tmp_center, 0.2f,
              new metal(
                  vector3(0.5f * (1.0 + hiprand_uniform(&local_rand_state)),
                          0.5f * (1.0 + hiprand_uniform(&local_rand_state)),
                          0.5f * (1.0 + hiprand_uniform(&local_rand_state))),
                  0.5f * hiprand_uniform(&local_rand_state)));
        } else {
          objs[i++] = new sphere(tmp_center, 0.2, new dielectric(1.5));
        }
      }
    }
    objs[i++] = new sphere(vector3(0, 1, 0), 1.0, new dielectric(1.5));
    objs[i++] = new sphere(vector3(-4, 1, 0), 1.0,
                           new lambertian(vector3(0.4, 0.2, 0.1)));
    objs[i++] = new sphere(vector3(4, 1, 0), 1.0,
                           new metal(vector3(0.7, 0.6, 0.5), 0.0));
    *rand_state = local_rand_state;
    *(tmp_scene) = new scene(objs, 22 * 22 + 1 + 3);
    vector3 lookfrom(13, 2, 3);
    vector3 lookat(0, 0, 0);
    float dist_to_focus = (lookfrom - lookat).length();
    float aperture = 0.1;
    *tmp_cam = new camera(lookfrom, lookat, vector3(0, 1, 0), 30.0,
                          float(nx) / float(ny), aperture, dist_to_focus);
  }
}
__global__ void free_scene(object **objs, scene **tmp_scene,
                           camera **d_camera) {
  delete *(objs);
  delete *(objs + 1);
  delete *(tmp_scene);
  delete *(d_camera);
}
int main() {
  int nx = 1920;
  int ny = 1080;
  int tx = 8;
  int ty = 8;
  int ray_num = 10;
  const int obj_nums = 22 * 22 + 1 + 3;
  std::cerr << "Rendering a " << nx << "x" << ny << " image ";
  std::cerr << "in " << tx << "x" << ty << " blocks.\n";

  int num_pixels = nx * ny;
  size_t fb_size = num_pixels * sizeof(vector3);
  clock_t start, stop;
  start = clock();
  // Render our buffer
  dim3 blocks(nx / tx + 1, ny / ty + 1);
  dim3 threads(tx, ty);
  // allocate FB
  vector3 *fb;
  checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

  // init scene

  scene **d_tmp_scene;
  checkCudaErrors(hipMalloc((void **)&d_tmp_scene, sizeof(scene *)));
  object **d_objs;
  checkCudaErrors(hipMalloc((void **)&d_objs, sizeof(object *) * obj_nums));
  hiprandState *d_rand_state;
  checkCudaErrors(
      hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));
  camera **d_tmp_cam;
  checkCudaErrors(hipMalloc((void **)&d_tmp_cam, sizeof(camera *)));
  rand_init<<<blocks, threads>>>(nx, ny, d_rand_state);

  init_scene<<<1, 1>>>(d_objs, d_tmp_scene, d_tmp_cam, nx, ny, d_rand_state);

  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  render<<<blocks, threads>>>(fb, nx, ny, ray_num, d_tmp_cam, d_tmp_scene,
                              d_rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  stop = clock();
  float timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
  std::cerr << nx << " * " << ny << std::endl;
  std::cerr << "took " << timer_seconds << " seconds.\n";

  // Output FB as Image
  pngmaster myImage(ny, nx);
  for (int j = ny - 1; j >= 0; j--) {
    for (int i = 0; i < nx; i++) {
      size_t pixel_index = j * nx + i;
      vector3 tmp_vec = fb[pixel_index] * 255.99f;
      myImage.set_pixel(i, j, tmp_vec.r(), tmp_vec.g(), tmp_vec.b());
    }
  }

  string file_name = "test" + std::to_string(timer_seconds) + ".png";
  myImage.output(file_name.c_str());
  std::cerr << "render finished" << std::endl;
  // free memory
  checkCudaErrors(hipDeviceSynchronize());
  free_scene<<<1, 1>>>(d_objs, d_tmp_scene, d_tmp_cam);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(d_objs));
  checkCudaErrors(hipFree(d_tmp_scene));
  checkCudaErrors(hipFree(fb));
  // useful for cuda-memcheck --leak-check full
  hipDeviceReset();
}